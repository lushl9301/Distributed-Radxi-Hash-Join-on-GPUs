
#include <hip/hip_runtime.h>
#include <cstdio>

/**check_cuda_error(char *, int )
* Function to identify the CUDA error using the error code.
* file  : the file where the error occured.
* line  : the line number where the error occured.
*/
void check_cuda_error(char *file, int line) {

  hipError_t error = hipGetLastError();

  if (error != hipSuccess) {
    printf("%s in %s at line %d", hipGetErrorString(error), file, line);
    exit(-1);
  }
}